#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mma.h>
#include <iostream>

#include "../include/utilities_CPU.h"
#include "../include/errchk_GPU.cuh"
#include "../include/matmul_wmma_GPU.cuh"

using namespace nvcuda;

__global__ void cuda_fma_wmma(float *A_, float *B_, float *C_, float *D,
                                   int N, int M, int P)
{
    // Define los fragmentos WMMA
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, float, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, float, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

    // Inicializar el fragmento del acumulador
    wmma::fill_fragment(c_frag, 0.0f);

    // Cargar los fragmentos de las matrices A y B
    wmma::load_matrix_sync(a_frag, a, MATRIX_SIZE);
    wmma::load_matrix_sync(b_frag, b, MATRIX_SIZE);

    // Realizar la operación FMA
    wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

    // Almacenar el resultado de vuelta en la matriz C
    wmma::store_matrix_sync(c, c_frag, MATRIX_SIZE, wmma::mem_row_major);
}

double __fma_wmma_GPU(float *A_, float *B_, float *C_, float *D,
                           int N, int M, int P)
{
    /**
     * Medición de tiempos
     */
    hipEvent_t start, stop;
    float exe_time_ms = 0.0;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    /**
     * Variables de mem. device
     */
    const size_t size_A = N * M * sizeof(float);
    const size_t size_B = M * P * sizeof(float);
    const size_t size_C = N * P * sizeof(float);
    float *d_A, *d_B, *d_C, *d_D;

    gpuErrchk(hipMalloc((void **)&d_A, size_A));
    gpuErrchk(hipMalloc((void **)&d_B, size_B));
    gpuErrchk(hipMalloc((void **)&d_C, size_C));
    gpuErrchk(hipMalloc((void **)&d_D, size_C));

    // Copiamos los datos necesarios para las matrices A, B y C
    gpuErrchk(hipMemcpy((void *)d_A, (const void *)A_, size_A, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy((void *)d_B, (const void *)B_, size_B, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy((void *)d_C, (const void *)C_, size_C, hipMemcpyHostToDevice));

    // Set execution configuration parameters
    //      threadsPerBlock: number of CUDA threads per grid block
    //      blocksPerGrid: number of blocks in grid
    dim3 threadsPerBlock(THR_PER_BLOCK, THR_PER_BLOCK);
    dim3 blocksPerGrid((P - 1) / threadsPerBlock.x + 1,
                       (N - 1) / threadsPerBlock.y + 1);

    // Launch kernel
    gpuErrchk(hipEventRecord(start));
    cuda_fma_wmma<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, d_D, N, M, P);
    cudaCheckError(); // Check error after execution
    gpuErrchk(hipEventRecord(stop));

    // Copy data from device array to host array
    gpuErrchk(hipMemcpy((void *)D, (const void *)d_D, size_C, hipMemcpyDeviceToHost));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&exe_time_ms, start, stop));

    /**
     * Free CUDA mem.
     */
    gpuErrchk(hipFree(d_A));
    gpuErrchk(hipFree(d_B));
    gpuErrchk(hipFree(d_C));
    gpuErrchk(hipFree(d_D));

    return (double)exe_time_ms;
}

double fma_wmma_GPU(float *A_, int N1, int M1,
                         float *B_, int N2, int M2,
                         float *C_, int N3, int M3,
                         float *D, int N, int M)
{
    if (!matrix_checkdims(N1, M1, N2, M2, N3, M3, N, M))
    {
        fprintf(stderr,
                "[DimError] La dimensiones de las matrices no coinciden: A(%d x %d) · B(%d x %d) + C(%d x %d) = D(%d x %d)\n",
                N1, M1, N2, M2, N3, M3, N, M);
        return 0.0; // Asum. que el checkeo no añade sobrecostes
    }

    return __fma_wmma_GPU(A_, B_, C_, D, N, M1, M);
}

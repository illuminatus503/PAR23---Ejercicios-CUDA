#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "../../include/cuda/fma.cuh"
#include "../../include/cuda/kernel_fma.cuh"
#include "../../include/cuda/kernel_cast.cuh"

#include "../../include/utils.h"
#include "../../include/cuda/error.cuh"

double fma_wmma_gpu_distrib(float *D, const float *A, const float *B, const float *C,
                            const int M, const int N, const int K,
                            const int M_split, const int N_split, const int K_split)
{
    hipEvent_t start, stop;
    float exe_time_ms = 0.0;

    float *A_sub, *B_sub, *C_sub, *D_sub;
    int i_size, j_size, k_size;

    int i, j, k;
    int i_sub, j_sub, k_sub;

    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    if (M_split <= 0)
    {
        perror("M_split is not positive!");
        exit(EXIT_FAILURE);
    }

    if (N_split <= 0)
    {
        perror("N_split is not positive!");
        exit(EXIT_FAILURE);
    }

    if (K_split <= 0)
    {
        perror("K_split is not positive!");
        exit(EXIT_FAILURE);
    }

    // Trasponemos B para matmul (accesos por filas en GPU)
    float *Bt = (float *)malloc(sizeof(float) * K * N);
    for (i = 0; i < K; i++)
    {
        for (j = 0; j < N; j++)
        {
            Bt[j * K + i] = B[i * N + j];
        }
    }

    // Calcular el tamaño de cada submatriz (considerando el padding si es necesario)
    int Msub = (M + M_split - 1) / M_split;
    int Nsub = (N + N_split - 1) / N_split;
    int Ksub = (K + K_split - 1) / K_split;

    // ! Reservamos buffers para los fragmentos de matrices
    float *d_C_sub;
    half *d_A_sub, *d_B_sub;
    float *d_A_sub_f32, *d_B_sub_f32;

    gpuErrchk(hipMalloc((void **)&d_A_sub_f32, Msub * Ksub * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&d_B_sub_f32, Ksub * Nsub * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&d_A_sub, Msub * Ksub * sizeof(half)));
    gpuErrchk(hipMalloc((void **)&d_B_sub, Ksub * Nsub * sizeof(half)));
    gpuErrchk(hipMalloc((void **)&d_C_sub, Msub * Nsub * sizeof(float)));

    // ! RUN distributed FMA with WMMA operations
    gpuErrchk(hipEventRecord(start));

    for (i = 0; i < M; i += Msub)
    {
        i_size = (i + Msub > M) ? M - i : Msub;

        for (j = 0; j < N; j += Nsub)
        {
            j_size = (j + Nsub > N) ? N - j : Nsub;

            // Copiamos datos de C a la memoria del device
            gpuErrchk(hipMalloc((void **)&d_C_sub, i_size * j_size * sizeof(float)));
            gpuErrchk(hipMemcpy((void *)d_C_sub, (const void *)(C + i * N + j), i_size * j_size * sizeof(float), hipMemcpyHostToDevice));

            // Agregamos las multiplicaciones de matrices sucesivas
            for (k = 0; k < K; k += Ksub)
            {
                k_size = (k + Ksub > K) ? K - k : Ksub;

                // Apuntar a las submatrices correspondientes de A y B
                A_sub = (float *)&(A[i * K + k]);
                B_sub = (float *)&(Bt[j * K + k]);

                // Enviar fragmentos de A y B_t (half) transpuesta al dispositivo
                gpuErrchk(hipMemcpy((void *)d_A_sub_f32, (const void *)A_sub, i_size * k_size * sizeof(half), hipMemcpyHostToDevice));
                gpuErrchk(hipMemcpy((void *)d_B_sub_f32, (const void *)B_sub, k_size * j_size * sizeof(half), hipMemcpyHostToDevice));

                // Configurar el tamaño del bloque y la cuadrícula para el kernel de conversión
                dim3 threadsPerBlockConv(256);
                dim3 blocksPerGridConv((i_size * k_size + threadsPerBlockConv.x - 1) / threadsPerBlockConv.x);

                // Lanzar el kernel de conversión para A
                f32_to_f16<<<blocksPerGridConv, threadsPerBlockConv>>>(d_A_sub, d_A_sub_f32, i_size * k_size);
                cudaCheckError();

                // Lanzar el kernel de conversión para B
                f32_to_f16<<<blocksPerGridConv, threadsPerBlockConv>>>(d_B_sub, d_B_sub_f32, k_size * j_size);
                cudaCheckError();

                gpuErrchk(hipDeviceSynchronize());

                // Realizar la operación FMA en las submatrices
                // Configuración del tamaño de bloque y cuadrícula para el kernel
                dim3 blockDim(WMMA_M, WMMA_N);
                dim3 gridDim((Msub + WMMA_M - 1) / WMMA_M, (Nsub + WMMA_N - 1) / WMMA_N);

                // Lanzamiento del kernel cuda_fma_wmma_ (B traspuesta)
                cuda_fma_wmma_rows<<<gridDim, blockDim>>>(d_C_sub, d_A_sub, d_B_sub, i_size, j_size, k_size, 1.0f, 1.0f);
                gpuErrchk(hipPeekAtLastError());
                gpuErrchk(hipDeviceSynchronize());
            }

            // Copiar los resultados de la multiplicación de matrices desde la GPU al host
            gpuErrchk(hipMemcpy((void *)(D + i * N + j), (const void *)d_C_sub, i_size * j_size * sizeof(float), hipMemcpyDeviceToHost));
        }
    }

    gpuErrchk(hipEventRecord(stop));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&exe_time_ms, start, stop));

    gpuErrchk(hipFree(d_A_sub));
    gpuErrchk(hipFree(d_B_sub));
    gpuErrchk(hipFree(d_A_sub_f32));
    gpuErrchk(hipFree(d_B_sub_f32));
    gpuErrchk(hipFree(d_C_sub));

    free(Bt);

    // // Calculamos el tamaño de las submatrices
    // int subM = (M + num_streams - 1) / num_streams; // cuantas matrices por filas
    // int subM_padded = (subM + WMMA_M - 1) / WMMA_M * WMMA_M; // Tamaño de submatriz con padding

    // const int N_padded = (N + WMMA_N - 1) / WMMA_N * WMMA_N;
    // const int K_padded = (K + WMMA_K - 1) / WMMA_K * WMMA_K;

    // // Reservar memoria para B una sola vez, ya que no cambia
    // half *d_B;
    // gpuErrchk(hipMalloc((void **)&d_B, K_padded * N_padded * sizeof(half)));

    // // Copiar y castear B a half con padding
    // half *B_padded = (half *)calloc(K_padded * N_padded, sizeof(half));
    // for (int i = 0; i < K; ++i)
    // {
    //     for (int j = 0; j < N; ++j)
    //     {
    //         B_padded[i * N_padded + j] = __float2half(B[i * N + j]);
    //     }
    // }
    // gpuErrchk(hipMemcpyAsync(d_B, B_padded, K_padded * N_padded * sizeof(half), hipMemcpyHostToDevice, streams[0]));
    // free(B_padded); // Liberar memoria del host

    // // Reservar memoria para d_A_sub, d_C_sub y d_C_padded_sub
    // half *d_A_sub;
    // float *d_C_sub, *d_C_padded_sub;
    // gpuErrchk(hipMalloc((void **)&d_A_sub, subM_padded * K_padded * sizeof(half)));
    // gpuErrchk(hipMalloc((void **)&d_C_sub, M * N * sizeof(float)));
    // gpuErrchk(hipMalloc((void **)&d_C_padded_sub, subM_padded * N_padded * sizeof(float)));

    // gpuErrchk(hipEventRecord(start));
    // for (int s = 0; s < num_streams; ++s)
    // {
    //     int offset_M = s * subM;
    //     int offset_M_padded = s * subM_padded;

    //     // Padding para A_sub y C_sub
    //     half *A_sub_padded = (half *)calloc(subM_padded * K_padded, sizeof(half));
    //     float *C_sub_padded = (float *)calloc(subM_padded * N_padded, sizeof(float));

    //     for (int i = 0; i < subM && (offset_M + i) < M; ++i)
    //     {
    //         for (int j = 0; j < K; ++j)
    //         {
    //             A_sub_padded[i * K_padded + j] = __float2half(A[(offset_M + i) * K + j]);
    //         }
    //         for (int j = 0; j < N; ++j)
    //         {
    //             C_sub_padded[i * N_padded + j] = C[(offset_M + i) * N + j];
    //         }
    //     }

    //     // Copiar datos al dispositivo en el stream actual
    //     gpuErrchk(hipMemcpyAsync(d_A_sub, A_sub_padded, subM_padded * K_padded * sizeof(half), hipMemcpyHostToDevice, streams[s]));
    //     gpuErrchk(hipMemcpyAsync(d_C_padded_sub, C_sub_padded, subM_padded * N_padded * sizeof(float), hipMemcpyHostToDevice, streams[s]));
    //     free(A_sub_padded); // Liberar memoria del host
    //     free(C_sub_padded); // Liberar memoria del host

    //     // Dimensiones del grid y del bloque para el stream actual
    //     dim3 blockDim(4 * WARP_SIZE, 4);
    //     dim3 gridDim((subM_padded + (WMMA_M * blockDim.x / WARP_SIZE - 1)) / (WMMA_M * blockDim.x / WARP_SIZE),
    //                  (N_padded + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y));

    //     // Lanzar kernel en el stream actual
    //     cuda_fma_wmma<<<gridDim, blockDim, 0, streams[s]>>>(d_C_padded_sub, d_B, d_A_sub, subM_padded, N_padded, K_padded, 1.0f, 1.0f);

    //     // Copiar los resultados de vuelta al host en el stream actual
    //     gpuErrchk(hipMemcpyAsync(d_C_sub + offset_M * N, d_C_padded_sub, subM * N * sizeof(float), hipMemcpyDeviceToHost, streams[s]));
    // }

    // gpuErrchk(hipEventRecord(stop));

    // // Esperar a que todos los streams completen su trabajo
    // for (int i = 0; i < num_streams; ++i)
    //     gpuErrchk(hipStreamSynchronize(streams[i]));

    // gpuErrchk(hipEventSynchronize(stop));
    // gpuErrchk(hipEventElapsedTime(&exe_time_ms, start, stop));

    // // Copiar los datos de C_sub a la matriz D original
    // gpuErrchk(hipMemcpy(D, d_C_sub, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // // Liberar recursos
    // gpuErrchk(hipFree(d_A_sub));
    // gpuErrchk(hipFree(d_B));
    // gpuErrchk(hipFree(d_C_sub));
    // gpuErrchk(hipFree(d_C_padded_sub));

    // // Liberar los streams
    // for (int i = 0; i < num_streams; ++i)
    //     gpuErrchk(hipStreamDestroy(streams[i]));

    return (double)exe_time_ms;
}

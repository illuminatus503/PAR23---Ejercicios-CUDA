#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "../../include/cuda/fma.cuh"
#include "../../include/cuda/kernel_fma.cuh"

#include "../../include/utils.h"
#include "../../include/cuda/error.cuh"

double fma_wmma_gpu_distrib(float *D, const int num_streams,
                            const float *A, const float *B, const float *C,
                            const int M, const int N, const int K)
{
    // hipEvent_t start, stop;
    // float exe_time_ms = 0.0;

    // gpuErrchk(hipEventCreate(&start));
    // gpuErrchk(hipEventCreate(&stop));

    // // Calculamos el tamaño de las submatrices
    // int subM = (M + num_streams - 1) / num_streams; // cuantas matrices por filas
    // int subM_padded = (subM + WMMA_M - 1) / WMMA_M * WMMA_M; // Tamaño de submatriz con padding

    // const int N_padded = (N + WMMA_N - 1) / WMMA_N * WMMA_N;
    // const int K_padded = (K + WMMA_K - 1) / WMMA_K * WMMA_K;

    // // Reservar memoria para B una sola vez, ya que no cambia
    // half *d_B;
    // gpuErrchk(hipMalloc((void **)&d_B, K_padded * N_padded * sizeof(half)));

    // // Copiar y castear B a half con padding
    // half *B_padded = (half *)calloc(K_padded * N_padded, sizeof(half));
    // for (int i = 0; i < K; ++i)
    // {
    //     for (int j = 0; j < N; ++j)
    //     {
    //         B_padded[i * N_padded + j] = __float2half(B[i * N + j]);
    //     }
    // }
    // gpuErrchk(hipMemcpyAsync(d_B, B_padded, K_padded * N_padded * sizeof(half), hipMemcpyHostToDevice, streams[0]));
    // free(B_padded); // Liberar memoria del host

    // // Reservar memoria para d_A_sub, d_C_sub y d_C_padded_sub
    // half *d_A_sub;
    // float *d_C_sub, *d_C_padded_sub;
    // gpuErrchk(hipMalloc((void **)&d_A_sub, subM_padded * K_padded * sizeof(half)));
    // gpuErrchk(hipMalloc((void **)&d_C_sub, M * N * sizeof(float)));
    // gpuErrchk(hipMalloc((void **)&d_C_padded_sub, subM_padded * N_padded * sizeof(float)));

    // gpuErrchk(hipEventRecord(start));
    // for (int s = 0; s < num_streams; ++s)
    // {
    //     int offset_M = s * subM;
    //     int offset_M_padded = s * subM_padded;

    //     // Padding para A_sub y C_sub
    //     half *A_sub_padded = (half *)calloc(subM_padded * K_padded, sizeof(half));
    //     float *C_sub_padded = (float *)calloc(subM_padded * N_padded, sizeof(float));

    //     for (int i = 0; i < subM && (offset_M + i) < M; ++i)
    //     {
    //         for (int j = 0; j < K; ++j)
    //         {
    //             A_sub_padded[i * K_padded + j] = __float2half(A[(offset_M + i) * K + j]);
    //         }
    //         for (int j = 0; j < N; ++j)
    //         {
    //             C_sub_padded[i * N_padded + j] = C[(offset_M + i) * N + j];
    //         }
    //     }

    //     // Copiar datos al dispositivo en el stream actual
    //     gpuErrchk(hipMemcpyAsync(d_A_sub, A_sub_padded, subM_padded * K_padded * sizeof(half), hipMemcpyHostToDevice, streams[s]));
    //     gpuErrchk(hipMemcpyAsync(d_C_padded_sub, C_sub_padded, subM_padded * N_padded * sizeof(float), hipMemcpyHostToDevice, streams[s]));
    //     free(A_sub_padded); // Liberar memoria del host
    //     free(C_sub_padded); // Liberar memoria del host

    //     // Dimensiones del grid y del bloque para el stream actual
    //     dim3 blockDim(4 * WARP_SIZE, 4);
    //     dim3 gridDim((subM_padded + (WMMA_M * blockDim.x / WARP_SIZE - 1)) / (WMMA_M * blockDim.x / WARP_SIZE),
    //                  (N_padded + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y));

    //     // Lanzar kernel en el stream actual
    //     cuda_fma_wmma<<<gridDim, blockDim, 0, streams[s]>>>(d_C_padded_sub, d_B, d_A_sub, subM_padded, N_padded, K_padded, 1.0f, 1.0f);

    //     // Copiar los resultados de vuelta al host en el stream actual
    //     gpuErrchk(hipMemcpyAsync(d_C_sub + offset_M * N, d_C_padded_sub, subM * N * sizeof(float), hipMemcpyDeviceToHost, streams[s]));
    // }

    // gpuErrchk(hipEventRecord(stop));

    // // Esperar a que todos los streams completen su trabajo
    // for (int i = 0; i < num_streams; ++i)
    //     gpuErrchk(hipStreamSynchronize(streams[i]));

    // gpuErrchk(hipEventSynchronize(stop));
    // gpuErrchk(hipEventElapsedTime(&exe_time_ms, start, stop));

    // // Copiar los datos de C_sub a la matriz D original
    // gpuErrchk(hipMemcpy(D, d_C_sub, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // // Liberar recursos
    // gpuErrchk(hipFree(d_A_sub));
    // gpuErrchk(hipFree(d_B));
    // gpuErrchk(hipFree(d_C_sub));
    // gpuErrchk(hipFree(d_C_padded_sub));

    // // Liberar los streams
    // for (int i = 0; i < num_streams; ++i)
    //     gpuErrchk(hipStreamDestroy(streams[i]));

    // return (double)exe_time_ms;
    return 0.0;
}

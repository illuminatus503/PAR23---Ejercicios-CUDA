#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "../../include/cuda/fma.cuh"
#include "../../include/cuda/kernel_fma.cuh"
#include "../../include/cuda/kernel_linalg.cuh"
#include "../../include/cuda/error.cuh"

double fma_wmma_gpu_distrib(float *D, float *A, float *B, float *C,
                            const int M, const int N, const int K,
                            const int M_split, const int N_split, const int K_split)
{
    hipEvent_t start, stop;
    float exe_time_ms = 0.0;

    int i, j, k;
    int i_size, j_size, k_size;
    int i_size_padded, j_size_padded, k_size_padded;

    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    if (M_split <= 0 || N_split <= 0 || K_split <= 0)
    {
        perror("M_split, N_split or K_split is not positive!");
        exit(EXIT_FAILURE);
    }

    // Calcular el tamaño de cada submatriz (considerando el padding si es necesario)
    // Ajustamos el tamaño de cada fragmento al padding que sea necesario, por dimensión,
    // para que sea múltiplo de 16.
    int Msub = (M + M_split - 1) / M_split;
    int max_i_size = (Msub + WMMA_M - 1) / WMMA_M * WMMA_M;
    int Nsub = (N + N_split - 1) / N_split;
    int max_j_size = (Nsub + WMMA_N - 1) / WMMA_N * WMMA_N;
    int Ksub = (K + K_split - 1) / K_split;
    int max_k_size = (Ksub + WMMA_K - 1) / WMMA_K * WMMA_K;

    // Reservamos buffers para los fragmentos de matrices en la GPU
    half *d_A_sub_f16, *d_B_sub_f16;
    float *d_A_sub_f32, *d_B_sub_f32, *d_C_sub;

    gpuErrchk(hipMalloc((void **)&d_A_sub_f32, max_i_size * max_k_size * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&d_B_sub_f32, max_k_size * max_j_size * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&d_A_sub_f16, max_i_size * max_k_size * sizeof(half)));
    gpuErrchk(hipMalloc((void **)&d_B_sub_f16, max_k_size * max_j_size * sizeof(half)));
    gpuErrchk(hipMalloc((void **)&d_C_sub, max_i_size * max_j_size * sizeof(float)));

    // RUN distributed FMA with WMMA operations
    gpuErrchk(hipEventRecord(start));

    for (i = 0; i < M; i += Msub)
    {
        // Calculamos la dimensión de filas del fragmento de D (padded)
        i_size = (i + Msub > M) ? M - i : Msub;
        i_size_padded = (i_size + WMMA_M - 1) / WMMA_M * WMMA_M;

        for (j = 0; j < N; j += Nsub)
        {
            // Calculamos la dimensión de columnas del fragmento de D (padded)
            j_size = (j + Nsub > N) ? N - j : Nsub;
            j_size_padded = (j_size + WMMA_N - 1) / WMMA_N * WMMA_N;

            // Inicializamos el fragmento C: copiamos solo los datos necesarios de C
            gpuErrchk(hipMemset((void *)d_C_sub, 0, max_i_size * max_j_size * sizeof(float)));
            gpuErrchk(hipMemcpy((void *)d_C_sub,
                                 (const void *)(C + i * Nsub + j),
                                 i_size * j_size * sizeof(float),
                                 hipMemcpyHostToDevice));

            for (k = 0; k < K; k += Ksub)
            {
                k_size = (k + Ksub > K) ? K - k : Ksub;
                k_size_padded = (k_size + WMMA_K - 1) / WMMA_K * WMMA_K;

                // Inicializamos los fragmentos A y Bt (con padding)
                gpuErrchk(hipMemset((void *)d_A_sub_f32, 0, max_i_size * max_k_size * sizeof(float)));
                gpuErrchk(hipMemset((void *)d_B_sub_f32, 0, max_k_size * max_j_size * sizeof(float)));
                gpuErrchk(hipMemcpy((void *)d_A_sub_f32,
                                     (const void *)(A + i * Ksub + k),
                                     i_size * k_size * sizeof(float),
                                     hipMemcpyHostToDevice));
                gpuErrchk(hipMemcpy((void *)d_B_sub_f32,
                                     (const void *)(B + k * Nsub + j),
                                     k_size * j_size * sizeof(float),
                                     hipMemcpyHostToDevice));

                // Convertimos los fragmentos A y B a half antes de operar
                f32_to_f16<<<256, (max_i_size * max_k_size + 256 - 1) / 256>>>(d_A_sub_f16, d_A_sub_f32, max_i_size * max_k_size);
                cudaCheckError();
                f32_to_f16<<<256, (max_k_size * max_j_size + 256 - 1) / 256>>>(d_B_sub_f16, d_B_sub_f32, max_k_size * max_j_size);
                cudaCheckError();
                gpuErrchk(hipDeviceSynchronize());

                // Realizamos la operación FMA en las submatrices
                dim3 blockDim(WMMA_M, WMMA_N);
                dim3 gridDim((i_size_padded + WMMA_M - 1) / WMMA_M,
                             (j_size_padded + WMMA_N - 1) / WMMA_N);
                cuda_fma_wmma<<<gridDim, blockDim>>>(d_C_sub,
                                                     d_A_sub_f16, d_B_sub_f16,
                                                     max_i_size, max_j_size, max_k_size,
                                                     1.0f, 1.0f);
                cudaCheckError();
                gpuErrchk(hipDeviceSynchronize());
            }

            // Copiamos los resultados de vuelta del dispositivo y extraemos el fragmento sin padding a D
            gpuErrchk(hipMemcpy((void *)(D + i * Nsub + j),
                                 (const void *)d_C_sub,
                                 i_size * j_size * sizeof(float),
                                 hipMemcpyDeviceToHost));
        }
    }

    gpuErrchk(hipEventRecord(stop));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&exe_time_ms, start, stop));

    // Free CUDA resources
    gpuErrchk(hipFree(d_A_sub_f16));
    gpuErrchk(hipFree(d_B_sub_f16));
    gpuErrchk(hipFree(d_A_sub_f32));
    gpuErrchk(hipFree(d_B_sub_f32));
    gpuErrchk(hipFree(d_C_sub));
    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));

    return (double)exe_time_ms;
}

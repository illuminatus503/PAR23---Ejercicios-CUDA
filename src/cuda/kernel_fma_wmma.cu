#include "hip/hip_runtime.h"
/**
 * El código de este kernel está basado en el código del artículo de nvidia.developer:
 *
 *      Programming Tensor Cores in CUDA 9 / Programmatic Access to Tensor Cores in CUDA 9.0 -->
 *      https://developer.nvidia.com/blog/programming-tensor-cores-cuda-9/#:~:text=Programmatic%20Access%20to%20Tensor%20Cores%20in%20CUDA%209.0
 *
 * Hay que compilar este kernel con soporte para SM 75 o superior. Por ejemplo, con -arch=sm_75.
 */

#include "../../include/cuda/kernel_fma.cuh"

#include <mma.h>
using namespace nvcuda; // compilar con

__global__ void cuda_fma_wmma(half *a, half *b, float *c,
                              int M, int N, int K,
                              float alpha, float beta)
{
    // Leading dimensions. Packed with no transpositions.
    int lda = M;
    int ldb = K;
    int ldc = M;

    // Tile using a 2D grid
    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

    // Declare the fragments
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    // Loop over k
    for (int i = 0; i < K; i += WMMA_K)
    {
        int aRow = warpM * WMMA_M;
        int aCol = i;

        int bRow = i;
        int bCol = warpN * WMMA_N;

        // Bounds checking
        if (aRow < M && aCol < K && bRow < K && bCol < N)
        {
            // Load the inputs
            wmma::load_matrix_sync(a_frag, a + aRow + aCol * lda, lda);
            wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

            // Perform the matrix multiplication
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }
    }

    // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
    int cRow = warpM * WMMA_M;
    int cCol = warpN * WMMA_N;

    if (cRow < M && cCol < N)
    {
        wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc, wmma::mem_col_major);

#pragma unroll
        for (int i = 0; i < c_frag.num_elements; i++)
        {
            c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
        }

        // Store the output
        wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc, wmma::mem_col_major);
    }
}

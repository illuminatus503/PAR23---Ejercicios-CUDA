#include "hip/hip_runtime.h"
/**
 * El código de este kernel está basado en el código del artículo de nvidia.developer:
 *
 *      Programming Tensor Cores in CUDA 9 / Programmatic Access to Tensor Cores in CUDA 9.0 -->
 *      https://developer.nvidia.com/blog/programming-tensor-cores-cuda-9/#:~:text=Programmatic%20Access%20to%20Tensor%20Cores%20in%20CUDA%209.0
 *
 *      WMMA_TensorCores_Examples (By WZSH)
 *      https://github.com/wzsh/wmma_tensorcore_sample/blob/master/matrix_wmma/matrix_wmma/main.cu
 *
 * Hay que compilar este kernel con soporte para SM 75 o superior. Por ejemplo, con -arch sm_75.
 */

#include <mma.h>
using namespace nvcuda; // compilar con

#include "../../include/cuda/kernel_fma.cuh"

__global__ void cuda_fma_wmma(half *A, half *B, float *C, float *D, int M, int N, int K)
{
    int a_col, a_row, b_col, b_row, c_col, c_row;
    int ix = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int iy = (blockIdx.y * blockDim.y + threadIdx.y);

    // Declare the fragments for A, B, accumulator for AB and C
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> ab_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

    // Initialize the AB fragment
    wmma::fill_fragment(ab_frag, 0.0f);

    // Perform AB = A*B
    a_row = ix * WMMA_M;
    b_row = iy * WMMA_N;
    for (int k = 0; k < K; k += WMMA_K)
    {
        a_col = k;
        b_col = k;

        if (a_row < M && a_col < K && b_row < K && b_col < N)
        {
            // Load the inputs
            wmma::load_matrix_sync(a_frag, A + a_col + a_row * K, K);
            wmma::load_matrix_sync(b_frag, B + b_col + b_row * K, K);

            // Perform the matrix multiplication
            wmma::mma_sync(ab_frag, a_frag, b_frag, ab_frag);
        }
    }

    // Perform D = AB + C
    c_col = b_row;
    c_row = a_row;
    if (c_row < M && c_col < N)
    {
        wmma::load_matrix_sync(c_frag, C + c_col + c_row * N, N, wmma::mem_row_major);

        for (int i = 0; i < c_frag.num_elements; i++)
        {
            c_frag.x[i] = ab_frag.x[i] + c_frag.x[i];
        }

        // Store the output
        wmma::store_matrix_sync(D + c_col + c_row * N, c_frag, N, wmma::mem_row_major);
    }
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../../include/cuda/error.cuh"
#include "../../include/cuda/kernel_linalg.cuh"
#include "../../include/cuda/linalg.cuh"

double transpose_cuda(float *out, float *in, const int M, const int N)
{
    // Eventos para medir el tiempo
    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    float milliseconds = 0;

    // Reserva de memoria para la matriz de entrada y salida en la GPU
    float *d_in, *d_out;
    gpuErrchk(hipMalloc((void **)&d_in, M * N * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&d_out, N * M * sizeof(float)));

    // Copiar la matriz de entrada al dispositivo
    gpuErrchk(hipMemcpy(d_in, in, M * N * sizeof(float), hipMemcpyHostToDevice));

    gpuErrchk(hipEventRecord(start));

    // Calcular dimensiones del grid y bloque para toda la matriz
    dim3 blockDim(TILE_DIM, BLOCK_ROWS);
    dim3 gridDim((N + TILE_DIM - 1) / TILE_DIM, (M + TILE_DIM - 1) / TILE_DIM);

    // Lanzamiento del kernel de transposición para toda la matriz
    cuda_transpose<<<gridDim, blockDim>>>(d_out, d_in, N, M); // Cambiar M por N y viceversa
    cudaCheckError();
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipEventRecord(stop));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&milliseconds, start, stop));

    // Copiar la matriz transpuesta de vuelta al host
    gpuErrchk(hipMemcpy(out, d_out, N * M * sizeof(float), hipMemcpyDeviceToHost));

    // Libera la memoria de las matrices en la GPU y destruye los eventos
    gpuErrchk(hipFree(d_in));
    gpuErrchk(hipFree(d_out));
    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));

    return (double)milliseconds;
}

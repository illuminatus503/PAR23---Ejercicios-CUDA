#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#include "../../include/cuda/fma.cuh"
#include "../../include/cuda/kernel_fma.cuh"

#include "../../include/utils.h"
#include "../../include/cuda/error.cuh"

double __fma_shared_gpu(float *A_, float *B_, float *C_, float *D,
                        int N, int M, int P)
{
    /**
     * Medición de tiempos
     */
    hipEvent_t start, stop;
    float exe_time_ms = 0.0;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    /**
     * Variables de mem. device
     */
    const size_t size_A = N * M * sizeof(float);
    const size_t size_B = M * P * sizeof(float);
    const size_t size_C = N * P * sizeof(float);
    float *d_A, *d_B, *d_C, *d_D;

    gpuErrchk(hipMalloc((void **)&d_A, size_A));
    gpuErrchk(hipMalloc((void **)&d_B, size_B));
    gpuErrchk(hipMalloc((void **)&d_C, size_C));
    gpuErrchk(hipMalloc((void **)&d_D, size_C));

    // Copiamos los datos necesarios para las matrices A, B y C
    gpuErrchk(hipMemcpy((void *)d_A, (const void *)A_, size_A, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy((void *)d_B, (const void *)B_, size_B, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy((void *)d_C, (const void *)C_, size_C, hipMemcpyHostToDevice));

    // Set execution configuration parameters
    //      threadsPerBlock: number of CUDA threads per grid block
    //      blocksPerGrid: number of blocks in grid
    dim3 threadsPerBlock(THR_PER_BLOCK, THR_PER_BLOCK);
    dim3 blocksPerGrid((P - 1) / threadsPerBlock.x + 1,
                       (N - 1) / threadsPerBlock.y + 1);

    // Launch kernel
    gpuErrchk(hipEventRecord(start));
    cuda_fma_sharedmem<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, d_D, N, M, P);
    cudaCheckError(); // Check error after execution
    gpuErrchk(hipEventRecord(stop));

    // Copy data from device array to host array
    gpuErrchk(hipMemcpy((void *)D, (const void *)d_D, size_C, hipMemcpyDeviceToHost));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&exe_time_ms, start, stop));

    /**
     * Free CUDA mem.
     */
    gpuErrchk(hipFree(d_A));
    gpuErrchk(hipFree(d_B));
    gpuErrchk(hipFree(d_C));
    gpuErrchk(hipFree(d_D));

    return (double)exe_time_ms;
}

double fma_shared_gpu(float *A_, int N1, int M1,
                      float *B_, int N2, int M2,
                      float *C_, int N3, int M3,
                      float *D, int N, int M)
{
    if (!matrix_checkdims(N1, M1, N2, M2, N3, M3, N, M))
    {
        fprintf(stderr,
                "[DimError] La dimensiones de las matrices no coinciden: A(%d x %d) · B(%d x %d) + C(%d x %d) = D(%d x %d)\n",
                N1, M1, N2, M2, N3, M3, N, M);
        return 0.0; // Asum. que el checkeo no añade sobrecostes
    }

    return __fma_shared_gpu(A_, B_, C_, D, N, M1, M);
}

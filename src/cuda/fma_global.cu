#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#include "../../include/cuda/fma.cuh"
#include "../../include/cuda/kernel_fma.cuh"
#include "../../include/cuda/error.cuh"

double fma_gpu_global(float *D, const float *A, const float *B, const float *C,
                      const int M, const int N, const int K)
{
    hipEvent_t start, stop;
    float exe_time_ms = 0.0;
    float *d_A, *d_B, *d_C;

    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    // Reservamos memoria para las matrices en el dispositivo
    gpuErrchk(hipMalloc((void **)&d_A, M * K * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&d_B, K * N * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&d_C, M * N * sizeof(float)));

    // Copiamos los datos necesarios para la operación: matrices A, B y C
    gpuErrchk(hipMemcpy((void *)d_A, (const void *)A, M * K * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy((void *)d_B, (const void *)B, K * N * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy((void *)d_C, (const void *)C, M * N * sizeof(float), hipMemcpyHostToDevice));

    // Asegúrate de que el número de hilos por bloque no sea mayor que el máximo permitido
    dim3 threadsPerBlock(WARP_SIZE, WARP_SIZE); // 1024 threads per block
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch kernel
    gpuErrchk(hipEventRecord(start));
    cuda_fma_global<<<blocksPerGrid, threadsPerBlock>>>(d_C, d_A, d_B, M, N, K, 1.0f, 1.0f);
    cudaCheckError();
    gpuErrchk(hipEventRecord(stop));

    // Copy data from device array to host array
    gpuErrchk(hipMemcpy((void *)D, (const void *)d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&exe_time_ms, start, stop));

    // Free CUDA resources
    gpuErrchk(hipFree(d_A));
    gpuErrchk(hipFree(d_B));
    gpuErrchk(hipFree(d_C));

    return (double)exe_time_ms;
}

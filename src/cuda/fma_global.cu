#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#include "../../include/cuda/fma.cuh"
#include "../../include/cuda/kernel_fma.cuh"

#include "../../include/utils.h"
#include "../../include/cuda/error.cuh"
#include "../../include/cuda/utils.cuh"

double fma_gpu_global(float *D, const float *A, const float *B, const float *C,
                      const int M, const int N, const int K)
{
    hipEvent_t start, stop;
    float exe_time_ms = 0.0;
    float *d_A, *d_B, *d_C, *d_D;

    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    // Reservamos memoria para las matrices en el dispositivo
    gpuErrchk(hipMalloc((void **)&d_A, M * K * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&d_B, K * N * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&d_C, M * N * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&d_D, M * N * sizeof(float)));

    // Copiamos los datos necesarios para la operación: matrices A, B y C
    gpuErrchk(hipMemcpy((void *)d_A, (const void *)A, M * K * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy((void *)d_B, (const void *)B, K * N * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy((void *)d_C, (const void *)C, M * N * sizeof(float), hipMemcpyHostToDevice));

    // Asegúrate de que el número de hilos por bloque no sea mayor que el máximo permitido
    dim3 threadsPerBlock(THR_PER_BLOCK, THR_PER_BLOCK);
    printf("[+] Lanzando 1024 hilos en warp: (%u x %u x %u)\n",
           threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z);

    // Calcula el número de bloques necesarios para cubrir todas las operaciones
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch kernel
    gpuErrchk(hipEventRecord(start));
    cuda_fma_global<<<blocksPerGrid, threadsPerBlock>>>(d_D, d_A, d_B, d_C, M, N, K);
    cudaCheckError();
    gpuErrchk(hipEventRecord(stop));

    // Copy data from device array to host array
    gpuErrchk(hipMemcpy((void *)D, (const void *)d_D, M * N * sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&exe_time_ms, start, stop));

    // Free CUDA resources
    gpuErrchk(hipFree(d_A));
    gpuErrchk(hipFree(d_B));
    gpuErrchk(hipFree(d_C));
    gpuErrchk(hipFree(d_D));

    return (double)exe_time_ms;
}

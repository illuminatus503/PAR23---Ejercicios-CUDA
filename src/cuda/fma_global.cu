#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../../include/cuda/fma.cuh"
#include "../../include/cuda/kernel_fma.cuh"
#include "../../include/cuda/error.cuh"

double fma_gpu_global(float *D, float *A, float *B, float *C,
                      const int M, const int N, const int K)
{
    float *d_A, *d_B, *d_C;
    float exe_time_ms = 0.0;

#ifdef DEBUG
    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
#endif

    // Reservamos memoria para las matrices en el dispositivo
    gpuErrchk(hipMalloc((void **)&d_A, M * K * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&d_B, K * N * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&d_C, M * N * sizeof(float)));

    // Copiamos los datos necesarios para la operación: matrices A, B y C
    gpuErrchk(hipMemcpy((void *)d_A, (const void *)A, M * K * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy((void *)d_B, (const void *)B, K * N * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy((void *)d_C, (const void *)C, M * N * sizeof(float), hipMemcpyHostToDevice));

    // Definimos el layout
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x,
                 (M + blockDim.y - 1) / blockDim.y);

#ifdef DEBUG
    gpuErrchk(hipEventRecord(start));
#endif

    // Launch kernel
    cuda_fma_global<<<gridDim, blockDim>>>(d_C, d_A, d_B, M, N, K, 1.0f, 1.0f);
    cudaCheckError();

#ifdef DEBUG
    gpuErrchk(hipEventRecord(stop));
#else
    gpuErrchk(hipDeviceSynchronize());
#endif

    // Copy data from device array to host array
    gpuErrchk(hipMemcpy((void *)D, (const void *)d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

#ifdef DEBUG
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&exe_time_ms, start, stop));
    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));
#endif

    // Free CUDA resources
    gpuErrchk(hipFree(d_A));
    gpuErrchk(hipFree(d_B));
    gpuErrchk(hipFree(d_C));

    return (double)exe_time_ms;
}

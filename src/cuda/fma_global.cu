#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#include "../../include/cuda/fma.cuh"
#include "../../include/cuda/kernel_fma.cuh"
#include "../../include/cuda/error.cuh"

double fma_gpu_global(float *D, const float *A, const float *B, const float *C,
                      const int M, const int N, const int K)
{
    hipEvent_t start, stop;
    float exe_time_ms = 0.0;

    size_t free_mem, total_mem;
    dim3 gridDim, blockDim;

    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));

    // ! CUDA set device and check memory availability
    hipSetDevice(0);
    hipMemGetInfo(&free_mem, &total_mem);
    if (free_mem < ((M * K + K * N + M * N) * sizeof(float)))
    {
        fprintf(stderr, "[ERROR] Not enough memory available!\n");
        exit(1);
    }

    // ! CUDA global memory allocation
    // Declaramos las var. de memoria global
    float *d_C;
    const float *d_A, *d_B;

    // Reservamos mem. global
    gpuErrchk(hipMalloc((void **)&d_A, M * K * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&d_B, K * N * sizeof(float)));
    gpuErrchk(hipMalloc((void **)&d_C, M * N * sizeof(float)));

    // Copiamos los datos desde mem. principal
    gpuErrchk(hipMemcpy((void *)d_A, (const void *)A, M * K * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy((void *)d_B, (const void *)B, K * N * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy((void *)d_C, (const void *)C, M * N * sizeof(float), hipMemcpyHostToDevice));

    // ! CUDA layout design
    // Block dimension (in threads per dim)
    blockDim.y = WARP_SIZE;
    blockDim.x = WARP_SIZE;

    // Grid dimension (in blocks per dim)
    gridDim.y = (M + blockDim.y - 1) / blockDim.y;
    gridDim.x = (N + blockDim.x - 1) / blockDim.x;

    // ! CUDA launch kernel
    gpuErrchk(hipEventRecord(start));
    cuda_gemm_global<<<blockDim, gridDim>>>(d_C, d_A, d_B, M, N, K, 1.0, 1.0);
    cudaCheckError();

    // ! CUDA copy data to local mem.
    gpuErrchk(hipMemcpy((void *)D, (const void *)d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    gpuErrchk(hipEventRecord(stop));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&exe_time_ms, start, stop));

    // ! Free CUDA resources
    gpuErrchk(hipFree((void *)d_A));
    gpuErrchk(hipFree((void *)d_B));
    gpuErrchk(hipFree((void *)d_C));

    return (double)exe_time_ms;
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>

#include "../include/codeCPU.h"
#include "../include/codeGPU.cuh"

#define TILE_SIZE 16      // Tile de 16x16 hilos, por bloque
#define THR_PER_BLOCK 256 // Número de hilos por bloque

__global__ void cuda_matmul_global(float *A_, float *B_, float *C_, float *D,
                                   int N, int M, int P)
{
    int i, j, k;
    float sum;

    /**
     * Calculamos el índice de i (filas, dim. y)
     */
    i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= N)
        return;

    /**
     * Calculamos el índice de j (columnas, dim. x)
     */
    j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j >= P)
        return;

    /**
     * Calcula el producto escalar de la fila i de A, columna j de B y
     * le suma el valor Cij: el resultado se guarda en Dij.
     *                      Dij = Cij + Ai_ · B_j
     */
    sum = C_[i * P + j];
    for (k = 0; k < M; k++)
    {
        sum += A_[i * M + k] * B_[k * P + j];
    }

    D[i * P + j] = sum; // solo escribimos una vez en mem. global de device
}

__global__ void cuda_matmul_sharedmem(float *A_, float *B_, float *C_, float *D,
                                      int N, int M, int P)
{
    int i, j, k;
    int tile_, tile_i, tile_j;
    float sum;

    // Inicializamos los tiles de A_ y B_
    __shared__ float A_shared[TILE_SIZE][TILE_SIZE];
    __shared__ float B_shared[TILE_SIZE][TILE_SIZE];

    // Calculamos los índices i, j de la matriz D
    i = blockIdx.y * blockDim.y + threadIdx.y;
    j = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculamos la suma
    sum = C_[i * P + j];
    for (tile_ = 0; tile_ < (M - 1) / TILE_SIZE + 1; tile_++)
    {
        // Load de la submatriz A_shared
        tile_j = tile_ * TILE_SIZE + threadIdx.x;
        if (i < N && tile_j < M)
        {
            A_shared[threadIdx.y][threadIdx.x] = A_[i * M + tile_j];
        }
        else
        {
            A_shared[threadIdx.y][threadIdx.x] = 0.0;
        }

        // Load de la submatriz B_shared
        tile_i = tile_ * TILE_SIZE + threadIdx.y;
        if (tile_i < M && j < P)
        {
            B_shared[threadIdx.y][threadIdx.x] = B_[tile_i * P + j];
        }
        else
        {
            B_shared[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads();

        for (k = 0; k < TILE_SIZE; k++)
        {
            sum += A_shared[threadIdx.y][k] * B_shared[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Escritura en mem. global de device (una sola vez)
    if (i < N && j < P)
    {
        D[i * P + j] = sum;
    }
}

double __fmadd_GPU(float *A_, float *B_, float *C_, float *D,
                   int N, int M, int P)
{
    /**
     * Medición de tiempos
     */
    hipEvent_t start, stop;
    float exe_time_ms = 0.0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /**
     * Variables de mem. device
     */
    const unsigned int size_A = N * M * sizeof(float);
    const unsigned int size_B = M * P * sizeof(float);
    const unsigned int size_C = N * P * sizeof(float);
    float *d_A, *d_B, *d_C, *d_D;

    gpuErrchk(hipMalloc((void **)&d_A, size_A));
    gpuErrchk(hipMalloc((void **)&d_B, size_B));
    gpuErrchk(hipMalloc((void **)&d_C, size_C));
    gpuErrchk(hipMalloc((void **)&d_D, size_C));

    // Copiamos los datos necesarios para las matrices A, B y C
    gpuErrchk(hipMemcpy(d_A, A_, size_A, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_B, B_, size_B, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_C, C_, size_C, hipMemcpyHostToDevice));

    // Set execution configuration parameters
    //      threadsPerBlock: number of CUDA threads per grid block
    //      blocksPerGrid: number of blocks in grid
    dim3 threadsPerBlock(THR_PER_BLOCK, THR_PER_BLOCK);
    dim3 blocksPerGrid((P - 1) / threadsPerBlock.x + 1,
                       (N - 1) / threadsPerBlock.y + 1);

    // Launch kernel
    gpuErrchk(hipEventRecord(start));
    cuda_matmul_global<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, d_D, N, M, P);
    gpuErrchk(hipEventRecord(stop));

    // Copy data from device array to host array
    hipMemcpy(D, d_D, size_C, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&exe_time_ms, start, stop);

    /**
     * Free CUDA mem.
     */
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);

    return (double)exe_time_ms;
}

double fmadd_GPU(float *A_, int N1, int M1,
                 float *B_, int N2, int M2,
                 float *C_, int N3, int M3,
                 float *D, int N, int M)
{
    if (!matrix_checkdims(N1, M1, N2, M2, N3, M3, N, M))
    {
        fprintf(stderr,
                "[DimError] La dimensiones de las matrices no coinciden: A(%d x %d) · B(%d x %d) + C(%d x %d) = D(%d x %d)\n",
                N1, M1, N2, M2, N3, M3, N, M);
        return 0.0; // Asum. que el checkeo no añade sobrecostes
    }

    return __fmadd_GPU(A_, B_, C_, D, N, M1, M);
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>

#include <stdio.h>

#include "../include/codeGPU.cuh"

#define THR_PER_BLOCK 1024

__global__ void cuda_vec_add(float *A, float *B, float *C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

double add_vectors_GPU(float *A, float *B, float *C, size_t N)
{
    hipEvent_t start, stop;
    float *d_A, *d_B, *d_C;
    float milliseconds = 0;
    int thr_per_blk, blk_in_grid;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    gpuErrchk(hipMalloc(&d_A, N * sizeof(float)));
    gpuErrchk(hipMalloc(&d_B, N * sizeof(float)));
    gpuErrchk(hipMalloc(&d_C, N * sizeof(float)));

    // Copy data from host arrays A and B to device arrays d_A and d_B
    gpuErrchk(hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice));

    // Set execution configuration parameters
    //      thr_per_blk: number of CUDA threads per grid block
    //      blk_in_grid: number of blocks in grid
    thr_per_blk = THR_PER_BLOCK;
    blk_in_grid = ceil((float)N / thr_per_blk);

    // Launch kernel
    gpuErrchk(hipEventRecord(start));
    cuda_vec_add<<<blk_in_grid, thr_per_blk>>>(d_A, d_B, d_C, N);
    gpuErrchk(hipEventRecord(stop));

    // Copy data from device array d_C to host array C
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return (milliseconds);
}
